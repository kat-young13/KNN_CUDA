#include "hip/hip_runtime.h"
//============================================================================
// Name        : KNN_CUDA.cpp
// Author      : 
// Version     :
// Copyright   : 
// Description : Hello World in C++, Ansi-style
//============================================================================

#include <iostream>
#include "libarff/arff_parser.h"
#include "libarff/arff_data.h"
using namespace std;

__global__ void KNN(int *predictions, float *dataset, int k, int instance_count, int attribute_count, float *k_distances, float *k_classes, int class_count)
{
    int tid = blockDim.x * blockIdx.x + threadIdx.x;

    float largest_array_distance = 0;
    int index_largest_distance;

    //for loop to look at other instances in array
    // j is the instance being compared to.
    int comp_cnt = 0;
    for(int j = 0; j < instance_count; j++){
    	if(j == tid) continue;

    	float distance = 0;

    	// CALCULATE DISTANCE BETWEEN J INSTANCE AND TID INSTANCE via ATTRIBUTES "D"
    	for(int d = 0; d < attribute_count - 1; d++){ // compute distance between two instances
    		float diff = dataset[tid * attribute_count + d] - dataset[j * attribute_count + d];
    		distance += diff * diff;
    	}
    	distance = sqrt(distance);

    	// PLACE DISTANCES AND CLASSES INTO ARRAYS FOR FIRST 5 - 6 INSTANCES DEPENDING ON CASE
    	if(j <= k){
    		if(tid <= k && tid != k){					// FILLS THE CLASSES ARRAY WITH FIRST FIVE THAT ARENT ITSELF
    			k_distances[tid * k + comp_cnt] = distance;
    			k_classes[tid * k + comp_cnt] = dataset[tid * attribute_count + attribute_count - 1];
    			if(distance > largest_array_distance){
    				largest_array_distance = distance;
    				index_largest_distance = comp_cnt;
    			}
    			comp_cnt++;
    		}else{
    			k_distances[tid * k + j] = distance;
    			k_classes[tid * k + j] = dataset[tid * attribute_count + attribute_count - 1];
    			if(distance > largest_array_distance){
    			    largest_array_distance = distance;
    			    index_largest_distance = j;
    			 }
    		}
    	}

    }

}

int main(int argc, char *argv[])
{
    if(argc != 2)
    {
        cout << "Usage: ./main datasets/datasetFile.arff" << endl;
        exit(0);
    }

    // READ IN DATASET
    ArffParser parser(argv[1]);
    ArffData *dataset = parser.parse();

    // GET METRICS FROM DATASET
    int instance_count = dataset->num_instances();
    int attribute_count = dataset->num_attributes();
    int class_count = dataset->num_classes();

    // MAKE DATASET INTO 1D ARRAY
    float *h_dataset = (float *)malloc(instance_count * attribute_count * sizeof(float));
    int count = 0;
    for (int i = 0; i < instance_count; i++){
    	for (int j = 0; j < attribute_count; j++){
    		h_dataset[count] = dataset->get_instance(i)->get(j)->operator float();
    		count++;
    	}
    }

    // START CLOCK
    struct timespec start, end;
    clock_gettime(CLOCK_MONOTONIC_RAW, &start);

    // SET K
    int k = 5;

    // Allocate other host memory
    int *h_predictions = (int *)malloc(instance_count * sizeof(int));
    float* h_Kdist =(float*)calloc(k * instance_count, sizeof(float));
    float* h_Kclasses=(float*)calloc(k * instance_count, sizeof(float));

    // Allocate the device input vector A
    int *d_predictions;
    float *d_distance_calculations; /// maybe get rid of?
    int *d_arr_class_data;
    float *d_arr_data;
    float *d_Kdist;
    float* d_Kclasses;


    hipMalloc(&d_predictions, instance_count * sizeof(int));
    hipMalloc(&d_distance_calculations, instance_count *sizeof(float));
    hipMalloc(&d_arr_class_data, instance_count * sizeof(int));
    hipMalloc(&d_arr_data, instance_count * attribute_count * sizeof(float));
    hipMalloc(&d_Kdist, k * instance_count * sizeof(float));
    hipMalloc(&d_Kclasses, k * instance_count * sizeof(float));

    // Copy the host input vectors A and B in host memory to the device input vectors in
    hipMemcpy(d_predictions, h_predictions, instance_count * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_arr_data, h_dataset, instance_count * attribute_count * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_Kdist, h_Kdist, k * instance_count * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_Kclasses, h_Kclasses, k * instance_count * sizeof(float), hipMemcpyHostToDevice);

    // Launch the Vector Add CUDA Kernel
    int threadsPerBlock = 256;
    int blocksPerGrid = (instance_count + threadsPerBlock - 1) / threadsPerBlock;

    printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);

    // set k value (number of neighbors)
    KNN<<<blocksPerGrid, threadsPerBlock>>>(d_predictions, d_arr_data, k, instance_count, attribute_count, d_Kdist, d_Kclasses,class_count);

    hipMemcpy(h_predictions, d_predictions, instance_count * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(h_Kclasses, d_Kclasses, k* instance_count * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(h_Kdist, d_Kdist, k* instance_count * sizeof(float), hipMemcpyDeviceToHost);

    for(int i = 0; i < 336 * 5; i++){
    	cout << h_Kdist[i];
    }

    /*

    int* predictions = KNN(dataset, 5);
    int* confusionMatrix = computeConfusionMatrix(predictions, dataset);
    float accuracy = computeAccuracy(confusionMatrix, dataset);

    clock_gettime(CLOCK_MONOTONIC_RAW, &end);
    uint64_t diff = (1000000000L * (end.tv_sec - start.tv_sec) + end.tv_nsec - start.tv_nsec) / 1e6;

    printf("The KNN classifier for %lu instances required %llu ms CPU time, accuracy was %.4f\n", dataset->num_instances(), (long long unsigned int) diff, accuracy);
	*/
}

